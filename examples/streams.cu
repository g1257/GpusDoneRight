
#include <hip/hip_runtime.h>
extern "C" __global__ void initArray(int *g_data, int *factor, int num_iterations,int n)
{ 
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx>=n) return;

	for(int i=0;i<num_iterations;i++)
		g_data[idx] += *factor;	// non-coalesced on purpose, to burn time
}

